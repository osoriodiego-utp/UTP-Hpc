#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__
void gpu_matrix_mult(float *d_a, float *d_b, float *d_c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((col < k) && (row < m))
    {
        float sum = 0.0;
        for (int i = 0; i < n; i++)
        {
            sum += d_a[row * n + i] * d_b[i * k + col];
        }
        d_c[row * k + col] = sum;
    }
}

void SaveMatrixToCsvFile(float *matrix, int m, int n, char *fileName)
{
    FILE *f = fopen(fileName, "w");

    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            fprintf(f, "%f,", matrix[i * n + j]);
        }
        long currentPos = ftell(f);
        fseek(f, currentPos - 1, SEEK_SET);
        fprintf(f, "\n");
    }

    fclose(f);
}

int main(int argc, char** argv)
{
    /*
       Execute on CPU
       Matrix h_a = host matrix A, size: m*n
       Matrix h_b = host matrix B, size: n*k
       Matrix h_c = host matrix C, size: m*k, is result of h_a * h_b

       Execute on GPU
       Matrix d_a = device matrix A, size: m*n
       Matrix d_b = device matrix B, size: n*k
       Matrix d_c = device matrix C, size: m*k, is result of d_a * d_b
    */

    // Matrices sizes:
    int m = 1024;
    int n = 5012;
    int k = 1024;

    // Host memory allocation
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(m * n * sizeof(float)); // Host matrix A
    h_b = (float*)malloc(n * k * sizeof(float)); // Host matrix B

    // Random value initialization of host matrix A
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    // Random value initialization of host matrix B
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    printf("\nMatrix initalized.");

    // Device memory allocation
    float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, m * n * sizeof(float)); // Device matrix A
    hipMalloc((void **) &d_b, n * k * sizeof(float)); // Device matrix B
    hipMalloc((void **) &d_c, m * k * sizeof(float)); // Device matrix C (result)

    printf("\nMemory allocated in device");

    // Copy matrixes from host to device memory
    hipMemcpy(d_a, h_a, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*n*k, hipMemcpyHostToDevice);

    printf("\nData copyed to device");

    int block_size = 32;
    dim3 dimBlock(block_size, block_size, 1);
    dim3 dimGrid(ceil((float)k / block_size), ceil((float)m / block_size), 1);

    printf("\nBlocks asigned");

    // Launch Kernel
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
    hipDeviceSynchronize();

    printf("\nKernel launched.");

    // Copy result from device to host
    h_c = (float*)malloc(m * k * sizeof(float)); // Host matrix C (result)
    hipMemcpy(h_c, d_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);

    SaveMatrixToCsvFile(h_a, m, n, "matrix_a");
    SaveMatrixToCsvFile(h_b, n, k, "matrix_b");
    SaveMatrixToCsvFile(h_c, m, k, "matrix_c");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}

